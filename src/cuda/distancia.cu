#include "hip/hip_runtime.h"
#include "../cac3.h"

double RT = 6378390.00;


__global__ void distancia(const float* ploc_lon,const float* ploc_lat,const float* prec_lon,const float* prec_lat, float* distancia){


int id=threadIdx.x + blockDim.x * blockIdx.x;

    
    distancia=sin(pl_lat[i])*sin(pr_lat);
    
    distancia+=cos(pl_lat[i])*cos(ploc_lon)*cos(prec_lat)*cos(prec_lon);
    
    distancia+=cos(pl_lat[i])*sin(ploc_lon)*cos(prec_lat)*sin(prec_lon);
    
    distancia=acos(distancia);


}

