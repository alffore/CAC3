/*
 * utiles.cu
 *
 *  Created on: 07/09/2014
 *      Author: alfonso
 */

#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

void memoriaGPUUso(const char * smensaje);

/**
 *
 *
 * @see https://devtalk.nvidia.com/default/topic/487541/best-way-to-report-memory-consumption-in-cuda-/
 */
void memoriaGPUUso(const char * smensaje) {
	// show memory usage of GPU

	hipError_t cuda_status;
	size_t free_byte;

	size_t total_byte;

	cuda_status = hipMemGetInfo(&free_byte, &total_byte);

	if (hipSuccess != cuda_status) {

		printf("Error: hipMemGetInfo fails, %s \n",
				hipGetErrorString(cuda_status));

		//exit(1);

	}

	double free_db = (double) free_byte;

	double total_db = (double) total_byte;

	double used_db = total_db - free_db;

	printf("%s ::: GPU memory usage: used = %f, free = %f MB, total = %f MB\n",smensaje,
			used_db / 1024.0 / 1024.0, free_db / 1024.0 / 1024.0,
			total_db / 1024.0 / 1024.0);
}
