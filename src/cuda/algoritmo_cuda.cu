#include "hip/hip_runtime.h"
/*
 * algoritmo_cuda.cu
 *
 *  Created on: 31/08/2014
 *      Author: alfonso
 */

#include "cac3.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "utils.h"

// Alojamiento de punteros en el dispositivo

//coordenadas geograficas de las localidades
float *d_lon_loc = NULL;
float *d_lat_loc = NULL;

//coordenadas geograficas de los recursos
float *d_lon_rec = NULL;
float *d_lat_rec = NULL;

//id de los recursos
unsigned int *d_id_rec = NULL;

//resultados del calculo
float *d_dist_rl = NULL;
unsigned int *d_id_rl = NULL;



void alojaMemoria_CLyRes_D(float* h_lon_loc, float* h_lat_loc,
		float * h_dist_rl,const size_t cuentaLocR);
void alojaMemoriaCR_D(float* h_lon_rec, float* h_lat_rec,
		unsigned int *h_id_rec, size_t cuentaRecT, float* h_dist_rl,
		const size_t cuentaLocR);

void liberaMemoria_CLyRes_D(void);
void liberaMemoriaCR_D(void);

void iniciaCalculo_v2(float *h_dist_rl, unsigned int *h_id_rl,
		const size_t cuentaRecT, const size_t cuentaLocR);

__global__ void calculaDKSM_v2(const float *d_lon_loc, const float *d_lat_loc,
		const float *d_lon_rec, const float *d_lat_rec,
		const unsigned int *d_id_rec, float *d_dist_rl, unsigned int *d_id_rl,
		const int cuentaLocR, const size_t maxall, const size_t offset);

__device__ float calculaDistancia(float lon0, float lat0, float lon1,
		float lat1);

/**
 *
 */
void alojaMemoria_CLyRes_D(float * h_lon_loc, float *h_lat_loc,
		float *h_dist_rl,const size_t cuentaLocR) {

	//coordenadas de Localidades
	hipMalloc((void**) &d_lon_loc, sizeof(float) * cuentaLocR);
	hipMalloc((void**) &d_lat_loc, sizeof(float) * cuentaLocR);

	//resultados
	hipMalloc((void**) &d_dist_rl, sizeof(float) * cuentaLocR);
	hipMalloc((void**) &d_id_rl, sizeof(unsigned int) * cuentaLocR);

	//copia de informacion de localidades
	hipMemcpy(d_lon_loc, h_lon_loc, sizeof(float) * cuentaLocR,
			hipMemcpyHostToDevice);
	hipMemcpy(d_lat_loc, h_lat_loc, sizeof(float) * cuentaLocR,
			hipMemcpyHostToDevice);

}
/**
 *
 */
void liberaMemoria_CLyRes_D(void) {
	hipFree(d_lon_loc);
	hipFree(d_lat_loc);

	hipFree(d_id_rl);
	hipFree(d_dist_rl);
}
/**
 *
 */
void alojaMemoriaCR_D(float * h_lon_rec, float *h_lat_rec,
		unsigned int *h_id_rec, const size_t cuentaRecT, float* h_dist_rl,
		const size_t cuentaLocR) {

	hipMalloc((void**) &d_lon_rec, sizeof(float) * cuentaRecT);
	hipMalloc((void**) &d_lat_rec, sizeof(float) * cuentaRecT);
	hipMalloc((void**) &d_id_rec, sizeof(unsigned int) * cuentaRecT);

	hipMemcpy(d_lon_rec, h_lon_rec, sizeof(float) * cuentaRecT,
			hipMemcpyHostToDevice);
	hipMemcpy(d_lat_rec, h_lat_rec, sizeof(float) * cuentaRecT,
			hipMemcpyHostToDevice);
	hipMemcpy(d_id_rec, h_id_rec, sizeof(unsigned int) * cuentaRecT,
			hipMemcpyHostToDevice);

	//incializa las distancias para fijar una alcance maximo a minimizar
	for (int j = 0; j < cuentaLocR; j++) {
		*(h_dist_rl + j) = 100.0f;
	}

	//copia de informacion de distancias
	hipMemcpy(d_dist_rl, h_dist_rl, sizeof(float) * cuentaLocR,
			hipMemcpyHostToDevice);
}



/**
 *
 */
void liberaMemoriaCR_D(void) {
	hipFree(d_lon_rec);
	hipFree(d_lat_rec);
	hipFree(d_id_rec);
}

/**
 *
 */

void iniciaCalculo_v2(float *h_dist_rl, unsigned int *h_id_rl,
		const size_t cuentaRecT, const size_t cuentaLocR) {

	int threads = THREADS;
	int blocks = MIN(BLOCKS,(cuentaLocR+threads-1) / threads);

	int numr_sh = MIN(cuentaRecT,MAX_PREC);

	int mod = cuentaRecT / numr_sh;
	int res = cuentaRecT % numr_sh;
	int m = 0;

	if (BDEP)
		printf(
				"Threads: %d, Blocks: %d (%ld), SM(PRECs): %d, MOD: %d, RES: %d\n",
				threads, blocks, (cuentaLocR + threads - 1) / threads, numr_sh,
				mod, res);

	while (m < mod) {

		if (BDEP)
			printf("Paso: %d, offset: %d\n", m, m * numr_sh);

		calculaDKSM_v2<<<blocks, threads, sizeof(RecM) * numr_sh>>>(d_lon_loc,
				d_lat_loc, d_lon_rec, d_lat_rec, d_id_rec, d_dist_rl, d_id_rl,
				cuentaLocR, numr_sh, m * numr_sh);

		hipDeviceSynchronize();
		checkCudaErrors(hipGetLastError());
		m++;
	}

	if (res > 0) {
		if (BDEP)
			printf("Paso residuo por %d\n", res);

		calculaDKSM_v2<<<blocks, threads, sizeof(RecM) * res>>>(d_lon_loc,
				d_lat_loc, d_lon_rec, d_lat_rec, d_id_rec, d_dist_rl, d_id_rl,
				cuentaLocR, res, mod * numr_sh);

		hipDeviceSynchronize();
		checkCudaErrors(hipGetLastError());
	}

	//obtiene resultados
	hipMemcpy(h_dist_rl, d_dist_rl, sizeof(float) * cuentaLocR,
			hipMemcpyDeviceToHost);
	hipMemcpy(h_id_rl, d_id_rl, sizeof(unsigned int) * cuentaLocR,
			hipMemcpyDeviceToHost);
}

// Sección de Kernel del algoritmo
/**
 *
 */

__global__ void calculaDKSM_v2(const float *d_lon_loc, const float *d_lat_loc,
		const float *d_lon_rec, const float *d_lat_rec,
		const unsigned int *d_id_rec, float *d_dist_rl, unsigned int *d_id_rl,
		const int cuentaLocR, const size_t maxall, const size_t offset) {

	int myId = threadIdx.x + blockDim.x * blockIdx.x;

	if (myId > cuentaLocR)
		return;

	extern __shared__ RecM rec[];

	int i;

	/*if (myId < maxall) {
	 rec[myId].lon = *(d_lon_rec + myId + offset);
	 rec[myId].lat = *(d_lat_rec + myId + offset);
	 rec[myId].id = *(d_id_rec + myId + offset);
	 }*/

	if (myId == 0) {
		for (i = 0; i < maxall; i++) {
			rec[i].lon = *(d_lon_rec + i + offset);
			rec[i].lat = *(d_lat_rec + i + offset);
			rec[i].id = *(d_id_rec + i + offset);
		}
	}

	__syncthreads();

	while (myId < cuentaLocR) {

		for (i = 0; i < maxall; i++) {

			float daux = calculaDistancia(*(d_lon_loc + myId),
					*(d_lat_loc + myId), rec[i].lon, rec[i].lat);

			if (*(d_dist_rl + myId) > daux) {
				*(d_dist_rl + myId) = daux;
				*(d_id_rl + myId) = rec[i].id;
			}

		}

		myId += blockDim.x * gridDim.x;
	}

}

/**
 *
 */__device__ float calculaDistancia(float lon0, float lat0, float lon1,
		float lat1) {

	return acosf(
			sinf(lat0) * sinf(lat1)
					+ cosf(lat0) * cosf(lon0) * cosf(lat1) * cosf(lon1)
					+ cosf(lat0) * sinf(lon0) * cosf(lat1) * sinf(lon1));

}
