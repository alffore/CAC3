#include "hip/hip_runtime.h"
/*
 * algoritmo_cuda.cu
 *
 *  Created on: 31/08/2014
 *      Author: alfonso
 */

#include "cac3.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "utils.h"

// Alojamiento de punteros en el dispositivo

//coordenadas geograficas de las localidades
float *d_lon_loc = NULL;
float *d_lat_loc = NULL;

//coordenadas geograficas de los recursos
float *d_lon_rec = NULL;
float *d_lat_rec = NULL;

//id de los recursos
unsigned int *d_id_rec = NULL;

//resultados del calculo
float *d_dist_rl = NULL;
unsigned int *d_id_rl = NULL;



//cantidad de Localidades
extern int cuentaLoc;

void alojaMemoriaCL_D(float* h_lon_loc, float* h_lat_loc);
void alojaMemoriaCR_D(float* h_lon_rec, float* h_lat_rec,
		unsigned int *h_id_rec, size_t cuentaRecT);
void alojaMemoriaRes(void);
void liberaMemoriaCL_D(void);
void liberaMemoriaCR_D(void);
void liberaMemoriaRes(void);

void iniciaCalculo(float *h_dist_rl, unsigned int *h_id_rl,
		const size_t cuentaRecT);

__global__ void calculaDK(const float *d_lon_loc, const float *d_lat_loc,
		const float *d_lon_rec, const float *d_lat_rec,
		const unsigned int *d_id_rec, float *d_dist_rl, unsigned int *d_id_rl,
		const size_t cuentaRecT, const int cuentaLoc);



__device__ float calculaDistancia(float lon0, float lat0, float lon1,
		float lat1);

/**
 *
 */
void iniciaCalculo(float *h_dist_rl, unsigned int *h_id_rl,
		const size_t cuentaRecT) {

	const int maxThreadsPerBlock = MAX_THREADS_BLOCK;
	int threads = maxThreadsPerBlock;
	int blocks = (int) (cuentaLoc / maxThreadsPerBlock) + 1;

	if(BDEP)printf("Threads: %d, Blocks: %d\n",threads,blocks);

	/*calculaDK<<<blocks, threads>>>(d_lon_loc, d_lat_loc, d_lon_rec, d_lat_rec,
			d_id_rec, d_dist_rl, d_id_rl, cuentaRecT, cuentaLoc);

	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());*/


	//obtiene resultados
	hipMemcpy(h_dist_rl, d_dist_rl, sizeof(float) * cuentaLoc,
			hipMemcpyDeviceToHost);
	hipMemcpy(h_id_rl, d_id_rl, sizeof(unsigned int) * cuentaLoc,
			hipMemcpyDeviceToHost);
}

/**
 *
 */
void alojaMemoriaCL_D(float * h_lon_loc, float *h_lat_loc) {

	hipMalloc((void**) &d_lon_loc, sizeof(float) * cuentaLoc);
	hipMalloc((void**) &d_lat_loc, sizeof(float) * cuentaLoc);


	hipMemcpy(d_lon_loc, h_lon_loc, sizeof(float) * cuentaLoc,
			hipMemcpyHostToDevice);
	hipMemcpy(d_lat_loc, h_lat_loc, sizeof(float) * cuentaLoc,
			hipMemcpyHostToDevice);
}

/**
 *
 */
void alojaMemoriaRes(void) {
	hipMalloc((void**) &d_dist_rl, sizeof(float) * cuentaLoc);
	hipMalloc((void**) &d_id_rl, sizeof(unsigned int) * cuentaLoc);


}

/**
 *
 */
void alojaMemoriaCR_D(float * h_lon_rec, float *h_lat_rec,
		unsigned int *h_id_rec, size_t cuentaRecT) {

	hipMalloc((void**) &d_lon_rec, sizeof(float) * cuentaRecT);
	hipMalloc((void**) &d_lat_rec, sizeof(float) * cuentaRecT);
	hipMalloc((void**) &d_id_rec, sizeof(unsigned int) * cuentaRecT);

	hipMemcpy(d_lon_rec, h_lon_rec, sizeof(float) * cuentaRecT,
			hipMemcpyHostToDevice);
	hipMemcpy(d_lat_rec, h_lat_rec, sizeof(float) * cuentaRecT,
			hipMemcpyHostToDevice);
	hipMemcpy(d_id_rec, h_id_rec, sizeof(unsigned int) * cuentaRecT,
			hipMemcpyHostToDevice);
}

/**
 *
 */
void liberaMemoriaCL_D(void) {
	hipFree(d_lon_loc);
	hipFree(d_lat_loc);
}

void liberaMemoriaCR_D(void) {
	hipFree(d_lon_rec);
	hipFree(d_lat_rec);
	hipFree(d_id_rec);
}

void liberaMemoriaRes(void) {
	hipFree(d_id_rl);
	hipFree(d_dist_rl);

}

// Sección de Kernel del algoritmo
/**
 *
 */
__global__ void calculaDK(const float *d_lon_loc, const float *d_lat_loc,
		const float *d_lon_rec, const float *d_lat_rec,
		const unsigned int *d_id_rec, float *d_dist_rl, unsigned int *d_id_rl,
		const size_t cuentaRecT, const int cuentaLoc) {

	int myId = threadIdx.x + blockDim.x * blockIdx.x;

	if (myId > cuentaLoc)
		return;

	//inicializacion arranque de kernel
	*(d_dist_rl + myId) = calculaDistancia(*(d_lon_loc + myId), *(d_lat_loc + myId), *d_lon_rec, *d_lat_rec);
	*(d_id_rl + myId) = *d_id_rec;


	for (unsigned int i = 1; i < cuentaRecT; i++) {

		float daux = calculaDistancia(*(d_lon_loc + myId), *(d_lat_loc + myId), *(d_lon_rec + i), *(d_lat_rec + i));

		if (daux < *(d_dist_rl + myId)) {
			*(d_dist_rl + myId) = daux;
			*(d_id_rl + myId) = *(d_id_rec + i);
		}

	}

}

/**
 *
 */
__device__ float calculaDistancia(float lon0, float lat0, float lon1,
		float lat1) {


	 float daux = sinf(lat0) * sinf(lat1);
	daux += cosf(lat0) * cosf(lon0) * cosf(lat1) * cosf(lon1);
	daux += cosf(lat0) * sinf(lon0) * cosf(lat1) * sinf(lon1);
	daux = acosf(daux);

	return daux;

}
