#include "hip/hip_runtime.h"
/*
 * algoritmo_cuda.cu
 *
 *  Created on: 31/08/2014
 *      Author: alfonso
 */

#include "cac3.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "utils.h"

// Alojamiento de punteros en el dispositivo

//coordenadas geograficas de las localidades
float *d_lon_loc = NULL;
float *d_lat_loc = NULL;

//coordenadas geograficas de los recursos
float *d_lon_rec = NULL;
float *d_lat_rec = NULL;

//id de los recursos
unsigned int *d_id_rec = NULL;

//resultados del calculo
float *d_dist_rl = NULL;
unsigned int *d_id_rl = NULL;

//cantidad de Localidades
extern int cuentaLoc;

void alojaMemoriaCLyRes_D(float* h_lon_loc, float* h_lat_loc,
		float * h_dist_rl);
void alojaMemoriaCR_D(float* h_lon_rec, float* h_lat_rec,
		unsigned int *h_id_rec, size_t cuentaRecT, float* h_dist_rl);

void liberaMemoriaCLyRes_D(void);
void liberaMemoriaCR_D(void);

void iniciaCalculo_v2(float *h_dist_rl, unsigned int *h_id_rl,
		const size_t cuentaRecT);

__global__ void calculaDKSM(const float *d_lon_loc, const float *d_lat_loc,
		const float *d_lon_rec, const float *d_lat_rec,
		const unsigned int *d_id_rec, float *d_dist_rl, unsigned int *d_id_rl,
		const int cuentaLoc, const size_t maxall, const size_t offset);

__device__ float calculaDistancia(float lon0, float lat0, float lon1,
		float lat1);

/**
 *
 */

void iniciaCalculo_v2(float *h_dist_rl, unsigned int *h_id_rl,
		const size_t cuentaRecT) {

	int threads = THREADS;
	int blocks = MIN(BLOCKS,(cuentaLoc+threads-1) / threads);

	int numr_sh = MIN(cuentaRecT,MAX_PREC);

	int mod = cuentaRecT / numr_sh;
	int res = cuentaRecT % numr_sh;
	int m;

	if (BDEP)
		printf(
				"Threads: %d, Blocks: %d (%d), SM(PRECs): %d, MOD: %d, RES: %d\n",
				threads, blocks, (cuentaLoc + threads - 1) / threads, numr_sh,
				mod, res);

	for (m = 0; m < mod; m++) {

		if (BDEP)
			printf("Paso: %d, offset: %d\n", m, m * numr_sh);

		calculaDKSM<<<blocks, threads, sizeof(RecM) * numr_sh>>>(d_lon_loc,
				d_lat_loc, d_lon_rec, d_lat_rec, d_id_rec, d_dist_rl, d_id_rl,
				 cuentaLoc, numr_sh, m * numr_sh);

		hipDeviceSynchronize();
		checkCudaErrors(hipGetLastError());

	}

	if (res > 0) {
		if (BDEP)
			printf("Paso residuo por %d\n", res);

		calculaDKSM<<<blocks, threads, sizeof(RecM) * res>>>(d_lon_loc,
				d_lat_loc, d_lon_rec, d_lat_rec, d_id_rec, d_dist_rl, d_id_rl,
				 cuentaLoc, res, (m + 1) * numr_sh);

		hipDeviceSynchronize();
		checkCudaErrors(hipGetLastError());
	}

	//obtiene resultados
	hipMemcpy(h_dist_rl, d_dist_rl, sizeof(float) * cuentaLoc,
			hipMemcpyDeviceToHost);
	hipMemcpy(h_id_rl, d_id_rl, sizeof(unsigned int) * cuentaLoc,
			hipMemcpyDeviceToHost);
}

/**
 *
 */
void alojaMemoriaCLyRes_D(float * h_lon_loc, float *h_lat_loc,
		float *h_dist_rl) {

	//coordenadas de Localidades
	hipMalloc((void**) &d_lon_loc, sizeof(float) * cuentaLoc);
	hipMalloc((void**) &d_lat_loc, sizeof(float) * cuentaLoc);

	//resultados
	hipMalloc((void**) &d_dist_rl, sizeof(float) * cuentaLoc);
	hipMalloc((void**) &d_id_rl, sizeof(unsigned int) * cuentaLoc);

	//copia de informacion de localidades
	hipMemcpy(d_lon_loc, h_lon_loc, sizeof(float) * cuentaLoc,
			hipMemcpyHostToDevice);
	hipMemcpy(d_lat_loc, h_lat_loc, sizeof(float) * cuentaLoc,
			hipMemcpyHostToDevice);

}

/**
 *
 */
void alojaMemoriaCR_D(float * h_lon_rec, float *h_lat_rec,
		unsigned int *h_id_rec, size_t cuentaRecT, float* h_dist_rl) {

	hipMalloc((void**) &d_lon_rec, sizeof(float) * cuentaRecT);
	hipMalloc((void**) &d_lat_rec, sizeof(float) * cuentaRecT);
	hipMalloc((void**) &d_id_rec, sizeof(unsigned int) * cuentaRecT);

	hipMemcpy(d_lon_rec, h_lon_rec, sizeof(float) * cuentaRecT,
			hipMemcpyHostToDevice);
	hipMemcpy(d_lat_rec, h_lat_rec, sizeof(float) * cuentaRecT,
			hipMemcpyHostToDevice);
	hipMemcpy(d_id_rec, h_id_rec, sizeof(unsigned int) * cuentaRecT,
			hipMemcpyHostToDevice);

	//incializa las distancias para fijar una lacance maximo a minimizar
	for (int j = 0; j < cuentaLoc; j++) {
		*(h_dist_rl + j) = 100.0f;
	}

	//copia de informacion de distancias
	hipMemcpy(d_dist_rl, h_dist_rl, sizeof(float) * cuentaLoc,
			hipMemcpyHostToDevice);
}

/**
 *
 */
void liberaMemoriaCLyRes_D(void) {
	hipFree(d_lon_loc);
	hipFree(d_lat_loc);

	hipFree(d_id_rl);
	hipFree(d_dist_rl);
}

/**
 *
 */
void liberaMemoriaCR_D(void) {
	hipFree(d_lon_rec);
	hipFree(d_lat_rec);
	hipFree(d_id_rec);
}

// Sección de Kernel del algoritmo
/**
 *
 */

__global__ void calculaDKSM(const float *d_lon_loc, const float *d_lat_loc,
		const float *d_lon_rec, const float *d_lat_rec,
		const unsigned int *d_id_rec, float *d_dist_rl, unsigned int *d_id_rl,
		const int cuentaLoc, const size_t maxall, const size_t offset) {

	int myId = threadIdx.x + blockDim.x * blockIdx.x;

	if (myId > cuentaLoc)
		return;

	int min_id;
	float min_dist = *(d_dist_rl + myId);

	float daux;

	extern __shared__ RecM rec[];

	if (myId < maxall) {
		rec[myId].lon = *(d_lon_rec + myId + offset);
		rec[myId].lat = *(d_lat_rec + myId + offset);
		rec[myId].id = *(d_id_rec + myId + offset);
	}

	__syncthreads();

	while (myId < cuentaLoc) {
		for (int i = 0; i < maxall; i++) {

			daux = calculaDistancia(*(d_lon_loc + myId), *(d_lat_loc + myId),
					rec[i].lon, rec[i].lat);

			if (min_dist > daux) {
				min_dist = daux;
				min_id = rec[i].id;
			}

		}

		*(d_dist_rl + myId) = min_dist;
		*(d_id_rl + myId) = min_id;

		myId += blockDim.x * gridDim.x;
	}

}

/**
 *
 */__device__ float calculaDistancia(float lon0, float lat0, float lon1,
		float lat1) {

	return acosf(
			sinf(lat0) * sinf(lat1)
					+ cosf(lat0) * cosf(lon0) * cosf(lat1) * cosf(lon1)
					+ cosf(lat0) * sinf(lon0) * cosf(lat1) * sinf(lon1));

}
