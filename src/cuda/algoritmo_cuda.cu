#include "hip/hip_runtime.h"
/*
 * algoritmo_cuda.cu
 *
 *  Created on: 31/08/2014
 *      Author: alfonso
 */

#include "cac3.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "utils.h"

// Alojamiento de punteros en el dispositivo

//coordenadas geograficas de las localidades
float *d_lon_loc = NULL;
float *d_lat_loc = NULL;

//coordenadas geograficas de los recursos
float *d_lon_rec = NULL;
float *d_lat_rec = NULL;

//id de los recursos
unsigned int *d_id_rec = NULL;

//resultados del calculo
float *d_dist_rl = NULL;
unsigned int *d_id_rl = NULL;

//cantidad de Localidades
extern int cuentaLoc;

void alojaMemoriaCL_D(float* h_lon_loc, float* h_lat_loc);
void alojaMemoriaCR_D(float* h_lon_rec, float* h_lat_rec,
		unsigned int *h_id_rec, size_t cuentaRecT);
void alojaMemoriaRes(void);
void liberaMemoriaCL_D(void);
void liberaMemoriaCR_D(void);
void liberaMemoriaRes(void);

void iniciaCalculo(float *h_dist_rl, unsigned int *h_id_rl,
		const size_t cuentaRecT);

/*__global__ void calculaDK(const float *d_lon_loc, const float *d_lat_loc,
 const float *d_lon_rec, const float *d_lat_rec,
 const unsigned int *d_id_rec, float *d_dist_rl, unsigned int *d_id_rl,
 const size_t cuentaRecT, const int cuentaLoc);*/

__global__ void calculaDK2(const float *d_lon_loc, const float *d_lat_loc,
		const float *d_lon_rec, const float *d_lat_rec,
		const unsigned int *d_id_rec, float *d_dist_rl, unsigned int *d_id_rl,
		const size_t cuentaRecT, const int cuentaLoc);

__global__ void calculaDKSM(const float *d_lon_loc, const float *d_lat_loc,
		const float *d_lon_rec, const float *d_lat_rec,
		const unsigned int *d_id_rec, float *d_dist_rl, unsigned int *d_id_rl,
		const size_t cuentaRecT, const int cuentaLoc, const size_t maxall);

__global__ void calculaDKSM2(const float *d_lon_loc, const float *d_lat_loc,
		const float *d_lon_rec, const float *d_lat_rec,
		const unsigned int *d_id_rec, float *d_dist_rl, unsigned int *d_id_rl,
		const size_t cuentaRecT, const int cuentaLoc, const size_t maxall);

__device__ float calculaDistancia(float lon0, float lat0, float lon1,
		float lat1);

/**
 *
 */
void iniciaCalculo(float *h_dist_rl, unsigned int *h_id_rl,
		const size_t cuentaRecT) {

	//const int maxThreadsPerBlock = MAX_THREADS_BLOCK;
	int threads = 64; //maxThreadsPerBlock;
	//int blocks = (cuentaLoc + threads - 1)/ threads;
	int blocks = MIN(30,(cuentaLoc+threads-1) / threads);

	int numr_sh = MIN(cuentaRecT,threads);

	if (BDEP)
		printf("Threads: %d, Blocks: %d (%d)\n", threads, blocks,
				(cuentaLoc + threads - 1) / threads);

	/*calculaDK2<<<blocks, threads>>>(d_lon_loc, d_lat_loc, d_lon_rec, d_lat_rec,
	 d_id_rec, d_dist_rl, d_id_rl, cuentaRecT, cuentaLoc);*/

	/*calculaDK<<<blocks, threads>>>(d_lon_loc, d_lat_loc, d_lon_rec, d_lat_rec,
	 d_id_rec, d_dist_rl, d_id_rl, cuentaRecT, cuentaLoc);*/

	/*calculaDKSM<<<blocks, threads, sizeof(RecM) * numr_sh>>>(d_lon_loc,
	 d_lat_loc, d_lon_rec, d_lat_rec, d_id_rec, d_dist_rl, d_id_rl,
	 cuentaRecT, cuentaLoc, numr_sh);*/

	calculaDKSM2<<<blocks, threads, sizeof(RecM) * numr_sh>>>(d_lon_loc,
			d_lat_loc, d_lon_rec, d_lat_rec, d_id_rec, d_dist_rl, d_id_rl,
			cuentaRecT, cuentaLoc, numr_sh);

	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());

	//obtiene resultados
	hipMemcpy(h_dist_rl, d_dist_rl, sizeof(float) * cuentaLoc,
			hipMemcpyDeviceToHost);
	hipMemcpy(h_id_rl, d_id_rl, sizeof(unsigned int) * cuentaLoc,
			hipMemcpyDeviceToHost);
}

/**
 *
 */
void alojaMemoriaCL_D(float * h_lon_loc, float *h_lat_loc) {

	hipMalloc((void**) &d_lon_loc, sizeof(float) * cuentaLoc);
	hipMalloc((void**) &d_lat_loc, sizeof(float) * cuentaLoc);

	hipMemcpy(d_lon_loc, h_lon_loc, sizeof(float) * cuentaLoc,
			hipMemcpyHostToDevice);
	hipMemcpy(d_lat_loc, h_lat_loc, sizeof(float) * cuentaLoc,
			hipMemcpyHostToDevice);
}

/**
 *
 */
void alojaMemoriaRes() {
	hipMalloc((void**) &d_dist_rl, sizeof(float) * cuentaLoc);
	hipMalloc((void**) &d_id_rl, sizeof(unsigned int) * cuentaLoc);
}

/**
 *
 */
void alojaMemoriaCR_D(float * h_lon_rec, float *h_lat_rec,
		unsigned int *h_id_rec, size_t cuentaRecT) {

	hipMalloc((void**) &d_lon_rec, sizeof(float) * cuentaRecT);
	hipMalloc((void**) &d_lat_rec, sizeof(float) * cuentaRecT);
	hipMalloc((void**) &d_id_rec, sizeof(unsigned int) * cuentaRecT);

	hipMemcpy(d_lon_rec, h_lon_rec, sizeof(float) * cuentaRecT,
			hipMemcpyHostToDevice);
	hipMemcpy(d_lat_rec, h_lat_rec, sizeof(float) * cuentaRecT,
			hipMemcpyHostToDevice);
	hipMemcpy(d_id_rec, h_id_rec, sizeof(unsigned int) * cuentaRecT,
			hipMemcpyHostToDevice);
}

/**
 *
 */
void liberaMemoriaCL_D(void) {
	hipFree(d_lon_loc);
	hipFree(d_lat_loc);
}

void liberaMemoriaCR_D(void) {
	hipFree(d_lon_rec);
	hipFree(d_lat_rec);
	hipFree(d_id_rec);
}

void liberaMemoriaRes(void) {
	hipFree(d_id_rl);
	hipFree(d_dist_rl);

}

// Sección de Kernel del algoritmo
/**
 *
 */
/*__global__ void calculaDK(const float *d_lon_loc, const float *d_lat_loc,
 const float *d_lon_rec, const float *d_lat_rec,
 const unsigned int *d_id_rec, float *d_dist_rl, unsigned int *d_id_rl,
 const size_t cuentaRecT, const int cuentaLoc) {

 int myId = threadIdx.x + blockDim.x * blockIdx.x;

 if (myId > cuentaLoc)
 return;

 //inicialización arranque de kernel
 *(d_dist_rl + myId) = calculaDistancia(*(d_lon_loc + myId),
 *(d_lat_loc + myId), *d_lon_rec, *d_lat_rec);
 *(d_id_rl + myId) = *d_id_rec;

 for (unsigned int i = 1; i < cuentaRecT; i++) {

 float daux = calculaDistancia(*(d_lon_loc + myId), *(d_lat_loc + myId),
 *(d_lon_rec + i), *(d_lat_rec + i));

 if (daux < *(d_dist_rl + myId)) {
 *(d_dist_rl + myId) = daux;
 *(d_id_rl + myId) = *(d_id_rec + i);
 }

 }

 }
 */

/**
 * @brief Segundo kernel que recorre un loop del grid hasta
 */__global__ void calculaDK2(const float *d_lon_loc, const float *d_lat_loc,
		const float *d_lon_rec, const float *d_lat_rec,
		const unsigned int *d_id_rec, float *d_dist_rl, unsigned int *d_id_rl,
		const size_t cuentaRecT, const int cuentaLoc) {

	int myId = threadIdx.x + blockDim.x * blockIdx.x;

	while (myId < cuentaLoc) {

		//inicialización arranque de kernel
		*(d_dist_rl + myId) = calculaDistancia(*(d_lon_loc + myId),
				*(d_lat_loc + myId), *d_lon_rec, *d_lat_rec);
		*(d_id_rl + myId) = *d_id_rec;

		/*for (unsigned int i = 1; i < cuentaRecT; i++) {

		 float daux = calculaDistancia(*(d_lon_loc + myId),
		 *(d_lat_loc + myId), *(d_lon_rec + i), *(d_lat_rec + i));

		 if (daux < *(d_dist_rl + myId)) {
		 *(d_dist_rl + myId) = daux;
		 *(d_id_rl + myId) = *(d_id_rec + i);
		 }

		 }*/
		myId += blockDim.x * gridDim.x;
	}

}

/**
 *
 */

__global__ void calculaDKSM(const float *d_lon_loc, const float *d_lat_loc,
		const float *d_lon_rec, const float *d_lat_rec,
		const unsigned int *d_id_rec, float *d_dist_rl, unsigned int *d_id_rl,
		const size_t cuentaRecT, const int cuentaLoc, const size_t maxall) {

	int myId = threadIdx.x + blockDim.x * blockIdx.x;

	int min_id;
	float min_dist;
	int offset = 0;
	//float daux;

	if (myId > cuentaLoc)
		return;

	min_dist = calculaDistancia(*(d_lon_loc + myId), *(d_lat_loc + myId),
			*d_lon_rec, *d_lat_rec);
	min_id = *d_id_rec;

	extern __shared__ RecM rec[];

	while (offset < cuentaRecT) {

		if (myId < maxall) {
			rec[myId].lon = *(d_lon_rec + myId + offset);
			rec[myId].lat = *(d_lat_rec + myId + offset);
			rec[myId].id = *(d_id_rec + myId + offset);
		}
		__syncthreads();

		for (int i = 0; i < maxall; i++) {

			/*daux = calculaDistancia(*(d_lon_loc + myId), *(d_lat_loc + myId),
			 rec[i].lon, rec[i].lat);

			 if (min_dist > daux) {
			 min_dist = daux;
			 min_id = rec[i].id;
			 }*/

		}

		*(d_dist_rl + myId) = min_dist;
		*(d_id_rl + myId) = min_id;

		__syncthreads();
		offset += MIN(maxall,cuentaRecT-1-offset);
	}

}

__global__ void calculaDKSM2(const float *d_lon_loc, const float *d_lat_loc,
		const float *d_lon_rec, const float *d_lat_rec,
		const unsigned int *d_id_rec, float *d_dist_rl, unsigned int *d_id_rl,
		const size_t cuentaRecT, const int cuentaLoc, const size_t maxall) {

	int myId = threadIdx.x + blockDim.x * blockIdx.x;

	int min_id;
	float min_dist;

	float daux;

	if (myId > cuentaLoc)
		return;

	min_dist = calculaDistancia(*(d_lon_loc + myId), *(d_lat_loc + myId),
			*d_lon_rec, *d_lat_rec);
	min_id = *d_id_rec;

	extern __shared__ RecM rec[];

	if (myId < maxall) {
		rec[myId].lon = *(d_lon_rec + myId);
		rec[myId].lat = *(d_lat_rec + myId);
		rec[myId].id = *(d_id_rec + myId);
	}
	__syncthreads();

	while (myId < cuentaLoc) {
		for (int i = 0; i < maxall; i++) {

			daux = calculaDistancia(*(d_lon_loc + myId), *(d_lat_loc + myId),
					rec[i].lon, rec[i].lat);

			if (min_dist > daux) {
				min_dist = daux;
				min_id = rec[i].id;
			}

		}

		//if (min_dist < *(d_dist_rl + myId)) {
			*(d_dist_rl + myId) = min_dist;
			*(d_id_rl + myId) = min_id;
		//}

		myId += blockDim.x * gridDim.x;
	}

}
/**
 *
 */__device__ float calculaDistancia(float lon0, float lat0, float lon1,
		float lat1) {

	return acosf(
			sinf(lat0) * sinf(lat1)
					+ cosf(lat0) * cosf(lon0) * cosf(lat1) * cosf(lon1)
					+ cosf(lat0) * sinf(lon0) * cosf(lat1) * sinf(lon1));

}
