/*
 * algoritmo.cu
 *
 *  Created on: 31/08/2014
 *      Author: AAFR <alffore@gmail.com>
 */

#include "cac3.h"
#include "utils.h"
#include <string.h>
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

//coordenadas geograficas de las localidades
float *h_lon_loc = NULL;
float *h_lat_loc = NULL;
unsigned int *h_id_loc = NULL;

//coordenadas geograficas de los recursos
float *h_lon_rec = NULL;
float *h_lat_rec = NULL;

//id de los recursos
unsigned int *h_id_rec = NULL;

//resultados del calculo
float *h_dist_rl = NULL;
unsigned int *h_id_rl = NULL;

unsigned int cuentaRecT;

extern int cuentaLoc;
extern PLocalidad PLr;
extern PRecurso PRr;
extern PTipoRec PTr;

//funciones de interfaz con GPU
extern void alojaMemoriaCL_D(float* h_lon_loc, float* h_lat_loc);
extern void alojaMemoriaCR_D(float* h_lon_rec, float* h_lat_rec,
		unsigned int *h_id_rec, size_t cuentaRecT);
extern void alojaMemoriaRes(void);
extern void liberaMemoriaCL_D(void);
extern void liberaMemoriaCR_D(void);
extern void liberaMemoriaRes(void);

extern void iniciaCalculo(float *h_dist_rl, unsigned int *h_id_rl,
		const size_t cuentaRecT);

char nombrearchivo[] = "cac4_salida.sql";
extern void abreArchivoSSQL(char * snomarch);
extern void cierraArchivoSSQL(void);

extern void insertaRes(float *h_dist_rl, unsigned int *h_id_rl, char *stipo);


//rutina para chequeo de memoria
extern void memoriaGPUUso(char * smensaje);


int calculoSD(void);
void alojaMemoriaCopiaLoc(void);
void alojaMemoriaCopiaLoc_v2(void);
void alojaMemoriaCopiaRec(char* stipo);
void liberaMemoriaLoc(void);
void liberaMemoriaLoc_v2(void);
void liberaMemoriaRec(void);
int cuentaRecTipo(char *stipo);

/**
 *
 */
int calculoSD(void) {

	checkCudaErrors(hipSetDevice(0));
	hipDeviceReset();
	if(BDEP)memoriaGPUUso("memoria antes de todo");

	//aloja la memoria del host
	alojaMemoriaCopiaLoc_v2();

	//aloja y copia la memoria al dispositivo
	alojaMemoriaCL_D(h_lon_loc, h_lat_loc);
	alojaMemoriaRes();

	printf("tam PrecM: %d\n",sizeof(RecM));

	// para cada tipo de recurso se ejecuta un "kernel"
	PTipoRec pt = PTr;



	abreArchivoSSQL(nombrearchivo);
	while (pt != NULL) {

		alojaMemoriaCopiaRec(pt->stipo_infra);

		if (BDEP)
			printf("Tema: %s (%u)\n", pt->stipo_infra, cuentaRecT);

		alojaMemoriaCR_D(h_lon_rec, h_lat_rec, h_id_rec, cuentaRecT);

		//checamos memoria antes de ejecucion de kernel
			if(BDEP)memoriaGPUUso("memoria antes de kernels");


		//llamada a kernel
		iniciaCalculo(h_dist_rl, h_id_rl, cuentaRecT);

		//imprime resultados
		insertaRes(h_dist_rl, h_id_rl, pt->stipo_infra);

		liberaMemoriaCR_D();
		liberaMemoriaRec();

		pt = pt->Pnext;
	}
	cierraArchivoSSQL();

	if(BDEP)memoriaGPUUso("memoria despues de kernels");

	//liberamos memoria en el device
	liberaMemoriaRes();
	liberaMemoriaCL_D();

	//liberamos la memoria empleada host
	liberaMemoriaLoc_v2();

	hipDeviceReset();
	return 0;
}

/**
 * @brief Función que aloja la memoria necesaria para las coordenadas de las localidades, distancia e id del recurso seleccionado
 */
void alojaMemoriaCopiaLoc(void) {

	int i = 0;
	h_lon_loc = (float *) malloc(sizeof(float) * cuentaLoc);
	h_lat_loc = (float *) malloc(sizeof(float) * cuentaLoc);
	h_id_loc = (unsigned int *) malloc(sizeof(unsigned int) * cuentaLoc);

	//alojamos memoria para los resultados en el host
	h_id_rl = (unsigned int*) malloc(sizeof(unsigned int) * cuentaLoc);
	h_dist_rl = (float *) malloc(sizeof(float) * cuentaLoc);

	PLocalidad ploc = PLr;

	while (ploc != NULL) {

		*(h_lon_loc + i) = (float) ploc->lon;
		*(h_lat_loc + i) = (float) ploc->lat;
		*(h_id_loc + i) = ploc->id_loc;
		ploc = ploc->Pnext;
		i++;
	}

}

void alojaMemoriaCopiaLoc_v2(void) {

	int i = 0;

	hipHostAlloc((void**) &h_lon_loc, sizeof(float) * cuentaLoc,
			hipHostMallocDefault);
	hipHostAlloc((void**) &h_lat_loc, sizeof(float) * cuentaLoc,
			hipHostMallocDefault);

	h_id_loc = (unsigned int *) malloc(sizeof(unsigned int) * cuentaLoc);

	//alojamos memoria para los resultados en el host
	h_id_rl = (unsigned int*) malloc(sizeof(unsigned int) * cuentaLoc);
	h_dist_rl = (float *) malloc(sizeof(float) * cuentaLoc);

	PLocalidad ploc = PLr;

	while (ploc != NULL) {

		*(h_lon_loc + i) = (float) ploc->lon;
		*(h_lat_loc + i) = (float) ploc->lat;
		*(h_id_loc + i) = ploc->id_loc;
		ploc = ploc->Pnext;
		i++;
	}

}

/**
 * @brief Función que aloja la cantidad de memoria necesaria para los recursos de cierto tipo
 */
void alojaMemoriaCopiaRec(char *stipo) {

	int i = 0;

	cuentaRecT = cuentaRecTipo(stipo);

	h_lon_rec = (float *) malloc(sizeof(float) * cuentaRecT);
	h_lat_rec = (float *) malloc(sizeof(float) * cuentaRecT);

	h_id_rec = (unsigned int *) malloc(sizeof(unsigned int) * cuentaRecT);

	PRecurso pr = PRr;
	while (pr != NULL) {
		if (strcmp(pr->stipo_infra, stipo) == 0) {
			*(h_lon_rec + i) = (float) pr->lon;
			*(h_lat_rec + i) = (float) pr->lat;
			*(h_id_rec + i) = pr->id;
			i++;
		}
		pr = pr->Pnext;
	}
}

/**
 * @brief Función que cuenta la cantidad recurso de un tipo
 */
int cuentaRecTipo(char *stipo) {

	int cuenta = 0;

	PRecurso pr = PRr;
	while (pr != NULL) {
		if (strcmp(pr->stipo_infra, stipo) == 0) {
			cuenta++;
		}
		pr = pr->Pnext;
	}

	return cuenta;
}

/**
 * @brief Función que libera la memoria asociada a las localidadess y la utilizada en los calculos asi como los resultados
 */
void liberaMemoriaLoc(void) {

	free(h_lon_loc);
	free(h_lat_loc);
	free(h_id_loc);

	//libera memoria local de resultados
	free(h_id_rl);
	free(h_dist_rl);

}

void liberaMemoriaLoc_v2(void) {

	hipHostFree(h_lon_loc);
	hipHostFree(h_lat_loc);

	free(h_id_loc);

	//libera memoria local de resultados
	free(h_id_rl);
	free(h_dist_rl);

}

/**
 * @brief Funcion que libera la memoria utilizada en los recursos
 */
void liberaMemoriaRec(void) {
	if (h_lon_rec != NULL)
		free(h_lon_rec);
	if (h_lat_rec != NULL)
		free(h_lat_rec);
	if (h_id_rec != NULL)
		free(h_id_rec);
}

