/*
 * algoritmo.cu
 *
 *  Created on: 31/08/2014
 *      Author: AAFR <alffore@gmail.com>
 */

#include "cac3.h"
#include "utils.h"
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <hip/hip_runtime.h>

//coordenadas geograficas de las localidades
float *h_lon_loc = NULL;
float *h_lat_loc = NULL;
unsigned int *h_id_loc = NULL;

//coordenadas geograficas e id de los recursos
float *h_lon_rec = NULL;
float *h_lat_rec = NULL;
unsigned int *h_id_rec = NULL;

//resultados del calculo
float *h_dist_rl = NULL;
unsigned int *h_id_rl = NULL;

unsigned int cuentaRecT;

extern int cuentaLoc;
extern PLocalidad PLr;
extern PRecurso PRr;
extern PTipoRec PTr;

//funciones de interfaz con GPU
extern void alojaMemoria_CLyRes_D(float* h_lon_loc, float* h_lat_loc,
		float * h_dist_rl, const size_t cuentaLocR);
extern void alojaMemoriaCR_D(float* h_lon_rec, float* h_lat_rec,
		unsigned int *h_id_rec, size_t cuentaRecT, float* h_dist_rl,
		const size_t cuentaLocR);

extern void liberaMemoria_CLyRes_D(void);
extern void liberaMemoriaCR_D(void);

extern void iniciaCalculo_v2(float *h_dist_rl, unsigned int *h_id_rl,
		const size_t cuentaRecT, const size_t cuentaLocR);

char nombrearchivo[] = "cac4_salida";
extern void abreArchivoSSQL(const char * snomarch);
extern void cierraArchivoSSQL(void);

extern void insertaRes(float *h_dist_rl, unsigned int *h_id_rl, char *stipo);

//rutina para chequeo de memoria
extern void memoriaGPUUso(const char * smensaje);

int calculoSD(void);

void alojaMemoriaCopiaLoc_v3(const size_t cantidad, int offset);
void alojaMemoriaCopiaRec(char* stipo);

void liberaMemoriaLoc_v3(void);
void liberaMemoriaRec(void);
int cuentaRecTipo(char *stipo);

/**
 *
 */
int calculoSD(void) {

	checkCudaErrors(hipSetDevice(0));
	hipDeviceReset();
	if (BDEP)
		memoriaGPUUso("memoria antes de todo");

	//inicamos loop de objetos

	int pasosLoc = cuentaLoc / MAX_LOCS;
	int resLoc = cuentaLoc % MAX_LOCS;
	int p = 0;
	size_t cuentaLocR;
	int offset = 0;

	if (resLoc == 0)
		pasosLoc--;

	if (BDEP)
		printf("pasosLoc: %d, resLoc: %d\n", pasosLoc, resLoc);
	do {

		//determinamos paso y offeset
		if (resLoc > 0) {

			if (p == pasosLoc) {
				cuentaLocR = resLoc;
			} else {
				cuentaLocR = MAX_LOCS;
			}

			offset = p * MAX_LOCS;
		} else {
			cuentaLocR = MAX_LOCS;
			offset = p * MAX_LOCS;
		}
		if (BDEP)
			printf("p: %d, cuentaLocR: %ld, offset: %d\n", p, cuentaLocR,
					offset);

		//aloja la memoria del host
		alojaMemoriaCopiaLoc_v3(cuentaLocR, offset);

		//aloja y copia la memoria al dispositivo
		alojaMemoria_CLyRes_D(h_lon_loc, h_lat_loc, h_dist_rl, cuentaLocR);

		// para cada tipo de recurso se ejecuta un "kernel"
		PTipoRec pt = PTr;

		char * snombrea;
		snombrea = (char*) malloc(sizeof(char) * 20);
		sprintf(snombrea, "%s_%d.sql", nombrearchivo, p);
		abreArchivoSSQL(snombrea);

		while (pt != NULL) {

			 alojaMemoriaCopiaRec(pt->stipo_infra);

			 if (BDEP)
			 printf("\nTema: %s (%u)\n", pt->stipo_infra, cuentaRecT);

			 alojaMemoriaCR_D(h_lon_rec, h_lat_rec, h_id_rec, cuentaRecT,
			 h_dist_rl, cuentaLocR);

			 //checamos memoria antes de ejecucion de kernel
			 //if (BDEP)memoriaGPUUso("memoria antes de kernels");

			 //llamada a kernel
			  iniciaCalculo_v2(h_dist_rl, h_id_rl, cuentaRecT, cuentaLocR);

			 //imprime resultados
			  insertaRes(h_dist_rl, h_id_rl, pt->stipo_infra);

			 liberaMemoriaCR_D();
			 liberaMemoriaRec();

			pt = pt->Pnext;
		}

		cierraArchivoSSQL();
		free(snombrea);

		if (BDEP)
			memoriaGPUUso("memoria despues de kernels");

		//liberamos memoria en el device
		liberaMemoria_CLyRes_D();

		//liberamos la memoria empleada host
		liberaMemoriaLoc_v3();

		p++;

	} while (p <= pasosLoc);

	hipDeviceReset();
	checkCudaErrors(hipGetLastError());
	return 0;
}

/**
 * @brief Función que aloja la memoria necesaria para las coordenadas de las localidades, distancia e id del recurso seleccionado
 *
 */

void alojaMemoriaCopiaLoc_v3(const size_t cuentaLocR, int offset) {

	hipHostAlloc((void**) &h_lon_loc, sizeof(float) * cuentaLocR,
			hipHostMallocDefault);
	hipHostAlloc((void**) &h_lat_loc, sizeof(float) * cuentaLocR,
			hipHostMallocDefault);

	/*h_lon_loc=(float *)malloc(sizeof(float) * cuentaLocR);
	 h_lat_loc=(float *)malloc(sizeof(float) * cuentaLocR);*/

	h_id_loc = (unsigned int *) malloc(sizeof(unsigned int) * cuentaLocR);

	//alojamos memoria para los resultados en el host
	h_id_rl = (unsigned int*) malloc(sizeof(unsigned int) * cuentaLocR);
	h_dist_rl = (float *) malloc(sizeof(float) * cuentaLocR);

	PLocalidad ploc = PLr;
	int i = offset;
	int j = 0;
	while (ploc != NULL && j < cuentaLocR) {

		*(h_lon_loc + i) = (float) ploc->lon;
		*(h_lat_loc + i) = (float) ploc->lat;
		*(h_id_loc + i) = ploc->id_loc;
		j++;

		ploc = ploc->Pnext;
		i++;
	}

}

/**
 * @brief Función que libera la memoria asociada a las localidades y la utilizada en los calculos asi como los resultados
 */
void liberaMemoriaLoc_v3(void) {

	hipHostFree(h_lon_loc);
	hipHostFree(h_lat_loc);
	/*free(h_lon_loc);
	 free(h_lat_loc);*/

	free(h_id_loc);

	//libera memoria local de resultados
	free(h_id_rl);
	free(h_dist_rl);

}

/**
 * @brief Función que aloja la cantidad de memoria necesaria para los recursos de cierto tipo
 */
void alojaMemoriaCopiaRec(char *stipo) {

	int i = 0;

	cuentaRecT = cuentaRecTipo(stipo);

	hipHostAlloc((void**) &h_lon_rec, sizeof(float) * cuentaRecT,
			hipHostMallocDefault);
	hipHostAlloc((void**) &h_lat_rec, sizeof(float) * cuentaRecT,
			hipHostMallocDefault);

	hipHostAlloc((void**) &h_id_rec, sizeof(unsigned int) * cuentaRecT,
			hipHostMallocDefault);

	/*h_lon_rec = (float *) malloc(sizeof(float) * cuentaRecT);
	 h_lat_rec = (float *) malloc(sizeof(float) * cuentaRecT);

	 h_id_rec = (unsigned int *) malloc(sizeof(unsigned int) * cuentaRecT);*/

	PRecurso pr = PRr;
	while (pr != NULL) {
		if (strcmp(pr->stipo_infra, stipo) == 0) {
			*(h_lon_rec + i) = (float) pr->lon;
			*(h_lat_rec + i) = (float) pr->lat;
			*(h_id_rec + i) = pr->id;
			i++;
		}
		pr = pr->Pnext;
	}

}

/**
 * @brief Funcion que libera la memoria utilizada en los recursos
 */
void liberaMemoriaRec(void) {

	if (h_lon_rec != NULL)
		hipHostFree(h_lon_rec);
	if (h_lat_rec != NULL)
		hipHostFree(h_lat_rec);
	if (h_id_rec != NULL)
		hipHostFree(h_id_rec);

	/*if (h_lon_rec != NULL)
	 free(h_lon_rec);
	 if (h_lat_rec != NULL)
	 free(h_lat_rec);
	 if (h_id_rec != NULL)
	 free(h_id_rec);*/
}

/**
 * @brief Función que cuenta la cantidad recurso de un tipo
 */
int cuentaRecTipo(char *stipo) {

	int cuenta = 0;

	PRecurso pr = PRr;
	while (pr != NULL) {
		if (strcmp(pr->stipo_infra, stipo) == 0) {
			cuenta++;
		}
		pr = pr->Pnext;
	}

	return cuenta;
}

